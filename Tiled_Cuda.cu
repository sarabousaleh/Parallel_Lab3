
#include <hip/hip_runtime.h>
#include <iostream>

#define TILE_SIZE 32

__global__ void matrixMulTiled(float *A, float *B, float *C, int M, int N, int K) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    
    float sum = 0.0f;
    
    for (int t = 0; t < (N - 1) / TILE_SIZE + 1; ++t) {
        if (row < M && t * TILE_SIZE + tx < N) 
            As[ty][tx] = A[row * N + t * TILE_SIZE + tx];
        else 
            As[ty][tx] = 0.0f;

        if (col < K && t * TILE_SIZE + ty < N)
            Bs[ty][tx] = B[(t * TILE_SIZE + ty) * K + col];
        else
            Bs[ty][tx] = 0.0f;

        __syncthreads();
        
        for (int i = 0; i < TILE_SIZE; ++i)
            sum += As[ty][i] * Bs[i][tx];
        
        __syncthreads();
    }
    
    if (row < M && col < K) 
        C[row * K + col] = sum;
}

int main() {
    const int M = 1024;
    const int N = 1024;
    const int K = 1024;

    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    int size_A = M * N * sizeof(float);
    int size_B = N * K * sizeof(float);
    int size_C = M * K * sizeof(float);

    // Allocate host memory
    h_A = (float *)malloc(size_A);
    h_B = (float *)malloc(size_B);
    h_C = (float *)malloc(size_C);

    // Initialize host arrays
    for (int i = 0; i < M * N; ++i) h_A[i] = 1.0f;
    for (int i = 0; i < N * K; ++i) h_B[i] = 2.0f;

    // Allocate device memory
    hipMalloc((void **)&d_A, size_A);
    hipMalloc((void **)&d_B, size_B);
    hipMalloc((void **)&d_C, size_C);

    // Copy host data to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((K + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    // Launch kernel
    matrixMulTiled<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    // Free device and host memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;
}
